#include "hip/hip_runtime.h"
#include <malloc.h>
#include <stdlib.h>
#include <stdio.h>
#include <io.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void bubbleeven(gpuarray){
    int i = threadIdx.x;
    int swap;
    if(gpuarray[2*i] > gpuarray[2*i+1]){
        swap = gpuarray[2*i];
        gpuarray[2*i] = gpuarray[2*i+1];
        gpuarray[2*i+1] = swap;
    }
}

__global__ void bubbleodd(gpuarray){
    int i = threadIdx.x;
    int swap;
    if(gpuarray[2*i+1] > gpuarray[2*i+2]){
        swap = gpuarray[2*i+1];
        gpuarray[2*i+1] = gpuarray[2*i+2];
        gpuarray[2*i+2] = swap;
    }

}

int main(){
  int i,k;
  int n = 20;
  int size;
  size = n * sizeof(int);
  
  int* cpuarray = (int*)malloc(size); 	
  int* gpuarray;
  hipMalloc(gpuarray, size); 
  
  for(i = 0; i < n; i++)
    scanf("%d", &cpuarray[i]);
    
  hipMemcpy(gpuarray, cpuarray, size, hipMemcpyHostToDevice);
  
  
  for(k = 0; k <= n-2; k++)
    if(k % 2 == 0){
        bubbleeven <<< 1, (n/2 - 1) >>>(gpuarray);
    }
    else
    {
        bubleodd <<< 1, (n/2 - 2) >>>(gpuarray);
    }
    
  
  
  hipMemcpy(cpuarray, gpuarray, size, hipMemcpyDeviceToHost);
  hipFree(gpuarray);
  
  for(i = 0; i < n; i++)
    printf("%d ", cpuarray[i]);

}